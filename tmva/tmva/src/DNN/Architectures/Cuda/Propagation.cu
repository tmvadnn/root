#include "hip/hip_runtime.h"
// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the functions required for the forward and //
 // backward propagation of activations through a neural network //
 // for CUDA architectures.                                      //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"

namespace TMVA {
namespace DNN  {

//____________________________________________________________________________
template<>
void TCuda<float>::MultiplyTranspose(TCudaMatrix<float> &output,
                                     const TCudaMatrix<float> &input,
                                     const TCudaMatrix<float> &Weights)
{
   int m, n, k;
   k = input.GetNcols();
   m = input.GetNrows();
   n = Weights.GetNrows();
   float alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A * B^T)
   hipStream_t s = input.GetComputeStream();
   hipblasSetStream(input.GetCublasHandle(), s);
   hipblasSgemm(input.GetCublasHandle(),
               HIPBLAS_OP_N, HIPBLAS_OP_T,
               m, n, k, & alpha,
               input.GetDataPointer(), m,     // *A, lda
               Weights.GetDataPointer(), n,   // *B, ldb
               & beta,                        // beta
               output.GetDataPointer(), m);   // *C, ldc
   output.SetComputeStream(s);
}

//____________________________________________________________________________
template<>
void TCuda<double>::MultiplyTranspose(TCudaMatrix<double> &output,
                                      const TCudaMatrix<double> &input,
                                      const TCudaMatrix<double> &Weights)
{
   int m, n, k;
   k = input.GetNcols();
   m = input.GetNrows();
   n = Weights.GetNrows();
   double alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A * B^T)
   hipStream_t s = input.GetComputeStream();
   hipblasSetStream(input.GetCublasHandle(), s);
   hipblasDgemm(input.GetCublasHandle(),
               HIPBLAS_OP_N, HIPBLAS_OP_T,
               m, n, k, & alpha,
               input.GetDataPointer(), m,     // *A, lda
               Weights.GetDataPointer(), n,   // *B, ldb
               & beta,                        // beta
               output.GetDataPointer(), m);   // *C, ldc
   output.SetComputeStream(s);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::AddRowWise(TCudaMatrix<AFloat> &Weights,
                               const TCudaMatrix<AFloat> &theta)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(Weights);
   hipStream_t s = Weights.GetComputeStream();
   ::TMVA::DNN::Cuda::AddRowWise<<<gridDims, blockDims, 0, s>>>(
       Weights.GetDataPointer(),
       theta.GetDataPointer(),
       Weights.GetNrows(),
       Weights.GetNcols());
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Backward(TCudaMatrix<AFloat> & activation_gradients_backward,
                             TCudaMatrix<AFloat> & weight_gradients,
                             TCudaMatrix<AFloat> & bias_gradients,
                             TCudaMatrix<AFloat> & df,
                             const TCudaMatrix<AFloat> & activation_gradients,
                             const TCudaMatrix<AFloat> & weights,
                             const TCudaMatrix<AFloat> & activation_backward)
{
   // Compute element-wise product.
   TCuda<AFloat>::Hadamard(df, activation_gradients);

   // Activation gradients.
   if (activation_gradients_backward.GetNoElements() > 0) {
      TCuda<AFloat>::Multiply(activation_gradients_backward, df, weights);
   }

   // Weight gradients.
   if (weight_gradients.GetNoElements() > 0) {
      TCuda<AFloat>::TransposeMultiply(weight_gradients, df, activation_backward);
   }

   // Bias gradients.
   if (bias_gradients.GetNoElements() > 0) {
      TCuda<AFloat>::SumColumns(bias_gradients, df);
   }

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Copy(TCudaMatrix<AFloat> & B,
                             const TCudaMatrix<AFloat> & A)
{
   size_t m = B.GetNrows();
   size_t n = B.GetNcols();
   hipMemcpyAsync(B.GetDataPointer(), A.GetDataPointer(),
                   m * n * sizeof(AFloat), hipMemcpyDeviceToDevice, 0);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Copy(std::vector<TCudaMatrix<AFloat>> & B,
                             const std::vector<TCudaMatrix<AFloat>> & A)
{
   for (size_t i = 0; i < B.size(); ++i) {
      Copy(B[i], A[i]);
   }
}

//____________________________________________________________________________

inline bool isInteger(double x)
{
   return x == floor(x);
}

int calculateDimension(size_t imgDim, size_t fltDim, size_t padding, size_t stride)
{
   double dimension = ((imgDim - fltDim + 2 * padding) / stride) + 1;
   if (!isInteger(dimension)) {
      std::cout << "Not compatible hyper parameters" << std::endl;
      std::exit(EXIT_FAILURE);
   }

   return (size_t)dimension;
}

///////////////////////////////////////////////////////////////////////////////////
/// \brief A helper for image operations that rearranges image regions into
///        column vectors.
///
/// \param[out] A The output matrix. Each row corresponds to a receptive field.
/// \param[in] B The input matrix. Each row corresponds to a row in the image view.
/// \param[in] imgHeight The heigh of the input.
/// \param[in] imgWidth The output of the input.
/// \param[in] fltHeight Height of the kernel.
/// \param[in] fltWidth Width of the kernel.
/// \param[in] strideRows stride size in the horizontal dimension.
/// \param[in] strideCols stride size in the vertical dimension.
/// \param[in] zeroPaddingHeight The padding in the horizontal dimension.
/// \param[in] zeroPaddingWidth The padding in the vertical dimension.
///
/// This transformation allows us to express a 2D convolution as a matrix
/// multiplication. We can therefore harness the finely tuned GEMM
/// implementation of cuBLAS to achieve maximum performance. This function
/// can greatly speed-up propagation in TConvLayer.
///////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
void TCuda<AFloat>::Im2col(TCudaMatrix<AFloat> &A,
                           const TCudaMatrix<AFloat> &B,
                           size_t imgHeight,
                           size_t imgWidth,
                           size_t fltHeight,
                           size_t fltWidth,
                           size_t strideRows,
                           size_t strideCols,
                           size_t zeroPaddingHeight,
                           size_t zeroPaddingWidth)
{
   size_t depth = B.GetNrows();

   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();

   ::TMVA::DNN::Cuda::Im2Col<<<gridDims, blockDims, 0, s>>>(A.GetDataPointer(), B.GetDataPointer(), depth, imgHeight, imgWidth,
                                                            fltHeight, fltWidth, strideRows, strideCols,
                                                            zeroPaddingHeight, zeroPaddingWidth);


}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::RotateWeights(TCudaMatrix<AFloat> &A,
                                  const TCudaMatrix<AFloat> &B,
                                  size_t filterDepth,
                                  size_t filterHeight,
                                  size_t filterWidth,
                                  size_t numFilters)
{

}


//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::ConvLayerBackward(std::vector<TCudaMatrix<AFloat>> & activation_gradients_backward,
                                      TCudaMatrix<AFloat> & weight_gradients,
                                      TCudaMatrix<AFloat> & bias_gradients,
                                      std::vector<TCudaMatrix<AFloat>> & df,
                                      const std::vector<TCudaMatrix<AFloat>> & activation_gradients,
                                      const TCudaMatrix<AFloat> & weights,
                                      const std::vector<TCudaMatrix<AFloat>> & activation_backward,
                                      size_t batchSize,
                                      size_t inputHeight,
                                      size_t inputWidth,
                                      size_t depth,
                                      size_t height,
                                      size_t width,
                                      size_t filterDepth,
                                      size_t filterHeight,
                                      size_t filterWidth,
                                      size_t nLocalViews)
{


}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvActivationGradients(
                                    std::vector<TCudaMatrix<AFloat>> & activation_gradients_backward,
                                    std::vector<TCudaMatrix<AFloat>> & df,
                                    const TCudaMatrix<AFloat> & weights,
                                    size_t batchSize,
                                    size_t inputHeight,
                                    size_t inputWidth,
                                    size_t depth,
                                    size_t height,
                                    size_t width,
                                    size_t filterDepth,
                                    size_t filterHeight,
                                    size_t filterWidth)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvWeightGradients(TCudaMatrix<AFloat> & weight_gradients,
                                                 std::vector<TCudaMatrix<AFloat>> & df,
                                                 const std::vector<TCudaMatrix<AFloat>> & activations_backward,
                                                 size_t batchSize,
                                                 size_t inputHeight,
                                                 size_t inputWidth,
                                                 size_t depth,
                                                 size_t height,
                                                 size_t width,
                                                 size_t filterDepth,
                                                 size_t filterHeight,
                                                 size_t filterWidth,
                                                 size_t nLocalViews)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvBiasGradients(TCudaMatrix<AFloat> & bias_gradients,
                                               std::vector<TCudaMatrix<AFloat>> & df,
                                               size_t batchSize,
                                               size_t depth,
                                               size_t nLocalViews)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::AddConvBiases(TCudaMatrix<AFloat> &output,
                                  const TCudaMatrix<AFloat> &biases)
{

}


//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Downsample(TCudaMatrix<AFloat> &A,
                               TCudaMatrix<AFloat> &B,
                               const TCudaMatrix<AFloat> &C,
                               size_t imgHeight,
                               size_t imgWidth,
                               size_t fltHeight,
                               size_t fltWidth,
                               size_t strideRows,
                               size_t strideCols)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::MaxPoolLayerBackward(std::vector<TCudaMatrix<AFloat>> & activationGradientsBackward,
                                         const std::vector<TCudaMatrix<AFloat>> & activationGradients,
                                         const std::vector<TCudaMatrix<AFloat>> & indexMatrix,
                                         size_t batchSize,
                                         size_t depth,
                                         size_t nLocalViews)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Reshape(TCudaMatrix<AFloat> &A, const TCudaMatrix<AFloat> &B)
{
   //TODO    
}

//______________________________________________________________________________
template <typename AReal>
void TCuda<AReal>::Rearrange(std::vector<TCudaMatrix<AReal>> &out, const std::vector<TCudaMatrix<AReal>> &in)
{
   // B x T x D out --- T x B x D in*/
   size_t B = out.size();
   size_t T = out[0].GetNrows();
   size_t D = out[0].GetNcols();
   if ((T != in.size()) || (B != in[0].GetNrows()) 
       || (D != in[0].GetNcols())) {
      std::cout << "Incompatible Dimensions\n"
         << in.size() << "x" << in[0].GetNrows() << "x" << in[0].GetNcols() 
         << " --> " << B << "x" << T << "x" << D << "\n";
      return;
   }
   for (size_t i = 0; i < B; ++i) {
      for (size_t j = 0; j < T; ++j) {
         for (size_t k = 0; k < D; ++k) {
            out[i](j, k) = in[j](i, k);
         }
      }
   }
   return;
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Flatten(TCudaMatrix<AFloat> &A,
                            const std::vector<TCudaMatrix<AFloat>> &B,
                            size_t size,
                            size_t nRows,
                            size_t nCols)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Deflatten(std::vector<TCudaMatrix<AFloat>> &A,
                              const TCudaMatrix<AFloat> &B,
                              size_t index,
                              size_t nRows,
                              size_t nCols)
{

}

} // namespace DNN
} // namespace TMVA
