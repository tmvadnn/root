#include "hip/hip_runtime.h"
// @(#)root/tmva/tmva/dnn:$Id$
// Author: Simon Pfreundschuh 13/07/16

/*************************************************************************
 * Copyright (C) 2016, Simon Pfreundschuh                                *
 * All rights reserved.                                                  *
 *                                                                       *
 * For the licensing terms see $ROOTSYS/LICENSE.                         *
 * For the list of contributors see $ROOTSYS/README/CREDITS.             *
 *************************************************************************/

 //////////////////////////////////////////////////////////////////
 // Implementation of the functions required for the forward and //
 // backward propagation of activations through a neural network //
 // for CUDA architectures.                                      //
 //////////////////////////////////////////////////////////////////

#include "TMVA/DNN/Architectures/Cuda.h"
#include "TMVA/DNN/Architectures/Cuda/Device.h"
#include "Kernels.cuh"
#include <math.h>

namespace TMVA {
namespace DNN  {

//____________________________________________________________________________
template<>
void TCuda<float>::MultiplyTranspose(TCudaMatrix<float> &output,
                                     const TCudaMatrix<float> &input,
                                     const TCudaMatrix<float> &Weights)
{
   int m, n, k;
   k = input.GetNcols();
   m = input.GetNrows();
   n = Weights.GetNrows();
   float alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A * B^T)
   hipStream_t s = input.GetComputeStream();
   hipblasSetStream(input.GetCublasHandle(), s);
   hipblasSgemm(input.GetCublasHandle(),
               HIPBLAS_OP_N, HIPBLAS_OP_T,
               m, n, k, & alpha,
               input.GetDataPointer(), m,     // *A, lda
               Weights.GetDataPointer(), n,   // *B, ldb
               & beta,                        // beta
               output.GetDataPointer(), m);   // *C, ldc
   output.SetComputeStream(s);
}

//____________________________________________________________________________
template<>
void TCuda<double>::MultiplyTranspose(TCudaMatrix<double> &output,
                                      const TCudaMatrix<double> &input,
                                      const TCudaMatrix<double> &Weights)
{
   int m, n, k;
   k = input.GetNcols();
   m = input.GetNrows();
   n = Weights.GetNrows();
   double alpha = 1.0, beta = 0.0;

   // Compute C = beta * C + alpha * (A * B^T)
   hipStream_t s = input.GetComputeStream();
   hipblasSetStream(input.GetCublasHandle(), s);
   hipblasDgemm(input.GetCublasHandle(),
               HIPBLAS_OP_N, HIPBLAS_OP_T,
               m, n, k, & alpha,
               input.GetDataPointer(), m,     // *A, lda
               Weights.GetDataPointer(), n,   // *B, ldb
               & beta,                        // beta
               output.GetDataPointer(), m);   // *C, ldc
   output.SetComputeStream(s);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::AddRowWise(TCudaMatrix<AFloat> &Weights,
                               const TCudaMatrix<AFloat> &theta)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(Weights);
   hipStream_t s = Weights.GetComputeStream();
   ::TMVA::DNN::Cuda::AddRowWise<<<gridDims, blockDims, 0, s>>>(
       Weights.GetDataPointer(),
       theta.GetDataPointer(),
       Weights.GetNrows(),
       Weights.GetNcols());
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Backward(TCudaMatrix<AFloat> & activation_gradients_backward,
                             TCudaMatrix<AFloat> & weight_gradients,
                             TCudaMatrix<AFloat> & bias_gradients,
                             TCudaMatrix<AFloat> & df,
                             const TCudaMatrix<AFloat> & activation_gradients,
                             const TCudaMatrix<AFloat> & weights,
                             const TCudaMatrix<AFloat> & activation_backward)
{
   // Compute element-wise product.
   TCuda<AFloat>::Hadamard(df, activation_gradients);

   // Activation gradients.
   if (activation_gradients_backward.GetNoElements() > 0) {
      TCuda<AFloat>::Multiply(activation_gradients_backward, df, weights);
   }

   // Weight gradients.
   if (weight_gradients.GetNoElements() > 0) {
      TCuda<AFloat>::TransposeMultiply(weight_gradients, df, activation_backward);
   }

   // Bias gradients.
   if (bias_gradients.GetNoElements() > 0) {
      TCuda<AFloat>::SumColumns(bias_gradients, df);
   }

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Copy(TCudaMatrix<AFloat> & B,
                             const TCudaMatrix<AFloat> & A)
{
   size_t m = B.GetNrows();
   size_t n = B.GetNcols();
   hipMemcpyAsync(B.GetDataPointer(), A.GetDataPointer(),
                   m * n * sizeof(AFloat), hipMemcpyDeviceToDevice, 0);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Copy(std::vector<TCudaMatrix<AFloat>> & B,
                             const std::vector<TCudaMatrix<AFloat>> & A)
{
   for (size_t i = 0; i < B.size(); ++i) {
      Copy(B[i], A[i]);
   }
}

//____________________________________________________________________________

inline bool isInteger(double x)
{
   return x == floor(x);
}

int calculateDimension(size_t imgDim, size_t fltDim, size_t padding, size_t stride)
{
   double dimension = ((imgDim - fltDim + 2 * padding) / stride) + 1;
   if (!isInteger(dimension)) {
      std::cout << "Not compatible hyper parameters" << std::endl;
      std::exit(EXIT_FAILURE);
   }

   return (size_t)dimension;
}

///////////////////////////////////////////////////////////////////////////////////
/// \brief A helper for image operations that rearranges image regions into
///        column vectors.
///
/// \param[out] A The output matrix. Each row corresponds to a receptive field.
/// \param[in] B The input matrix. Each row corresponds to a row in the image view.
/// \param[in] imgHeight The heigh of the input.
/// \param[in] imgWidth The output of the input.
/// \param[in] fltHeight Height of the kernel.
/// \param[in] fltWidth Width of the kernel.
/// \param[in] strideRows stride size in the horizontal dimension.
/// \param[in] strideCols stride size in the vertical dimension.
/// \param[in] zeroPaddingHeight The padding in the horizontal dimension.
/// \param[in] zeroPaddingWidth The padding in the vertical dimension.
///
/// This transformation allows us to express a 2D convolution as a matrix
/// multiplication. We can therefore harness the finely tuned GEMM
/// implementation of cuBLAS to achieve maximum performance. This function
/// can greatly speed-up propagation in TConvLayer.
///////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
void TCuda<AFloat>::Im2col(TCudaMatrix<AFloat> &A,
                           const TCudaMatrix<AFloat> &B,
                           size_t imgHeight,
                           size_t imgWidth,
                           size_t fltHeight,
                           size_t fltWidth,
                           size_t strideRows,
                           size_t strideCols,
                           size_t zeroPaddingHeight,
                           size_t zeroPaddingWidth)
{
   size_t depth = B.GetNrows();

   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();

   ::TMVA::DNN::Cuda::Im2Col<<<gridDims, blockDims, 0, s>>>(A.GetDataPointer(), B.GetDataPointer(), depth, imgHeight, imgWidth,
                                                            fltHeight, fltWidth, strideRows, strideCols,
                                                            zeroPaddingHeight, zeroPaddingWidth);


}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::RotateWeights(TCudaMatrix<AFloat> &A,
                                  const TCudaMatrix<AFloat> &B,
                                  size_t filterDepth,
                                  size_t filterHeight,
                                  size_t filterWidth,
                                  size_t numFilters)
{

}


//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::ConvLayerBackward(std::vector<TCudaMatrix<AFloat>> & activation_gradients_backward,
                                      TCudaMatrix<AFloat> & weight_gradients,
                                      TCudaMatrix<AFloat> & bias_gradients,
                                      std::vector<TCudaMatrix<AFloat>> & df,
                                      const std::vector<TCudaMatrix<AFloat>> & activation_gradients,
                                      const TCudaMatrix<AFloat> & weights,
                                      const std::vector<TCudaMatrix<AFloat>> & activation_backward,
                                      size_t batchSize,
                                      size_t inputHeight,
                                      size_t inputWidth,
                                      size_t depth,
                                      size_t height,
                                      size_t width,
                                      size_t filterDepth,
                                      size_t filterHeight,
                                      size_t filterWidth,
                                      size_t nLocalViews)
{


}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvActivationGradients(
                                    std::vector<TCudaMatrix<AFloat>> & activation_gradients_backward,
                                    std::vector<TCudaMatrix<AFloat>> & df,
                                    const TCudaMatrix<AFloat> & weights,
                                    size_t batchSize,
                                    size_t inputHeight,
                                    size_t inputWidth,
                                    size_t depth,
                                    size_t height,
                                    size_t width,
                                    size_t filterDepth,
                                    size_t filterHeight,
                                    size_t filterWidth)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvWeightGradients(TCudaMatrix<AFloat> & weight_gradients,
                                                 std::vector<TCudaMatrix<AFloat>> & df,
                                                 const std::vector<TCudaMatrix<AFloat>> & activations_backward,
                                                 size_t batchSize,
                                                 size_t inputHeight,
                                                 size_t inputWidth,
                                                 size_t depth,
                                                 size_t height,
                                                 size_t width,
                                                 size_t filterDepth,
                                                 size_t filterHeight,
                                                 size_t filterWidth,
                                                 size_t nLocalViews)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::CalculateConvBiasGradients(TCudaMatrix<AFloat> & bias_gradients,
                                               std::vector<TCudaMatrix<AFloat>> & df,
                                               size_t batchSize,
                                               size_t depth,
                                               size_t nLocalViews)
{

}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::AddConvBiases(TCudaMatrix<AFloat> &output,
                                  const TCudaMatrix<AFloat> &biases)
{

}


//____________________________________________________________________________
//////////////////////////////////////////////////////////////////////////////////////////////
/// \brief Downsampling function used as the forward propagation step of a
///        Max-Pooling layer.
///
/// \param[out] A The output matrix. Each row corresponds to a slice and each element
///             is the max within a receptive field.
/// \param[out] B The winning indices matrix. Each element is the index of the max element.
/// \param[in] C The input matrix. Each row is a slice.
/// \param[in] imgHeight The heigh of the input.
/// \param[in] imgWidth The output of the input.
/// \param[in] fltHeight Height of the kernel.
/// \param[in] fltWidth Width of the kernel.
/// \param[in] strideRows stride size in the horizontal dimension.
/// \param[in] strideCols stride size in the vertical dimension.
///
/// Each output element is the maximum of the receptive field. We also save the winning
/// indices to facilitate back-propagation - we need to know which input element influenced
/// the output and only apply the derivative correction to this particular element.
/// The slicing process is the same as in a convolutional layer, however padding is set to 0.
///////////////////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
void TCuda<AFloat>::Downsample(TCudaMatrix<AFloat> &A,
                               TCudaMatrix<AFloat> &B,
                               const TCudaMatrix<AFloat> &C,
                               size_t imgHeight,
                               size_t imgWidth,
                               size_t fltHeight,
                               size_t fltWidth,
                               size_t strideRows,
                               size_t strideCols)
{
   size_t depth = C.GetNrows();

   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();
   ::TMVA::DNN::Cuda::Downsample<<<gridDims, blockDims, 0, s>>>(A.GetDataPointer(), B.GetDataPointer(),
                                                                C.GetDataPointer(), depth, imgHeight, imgWidth,
                                                                fltHeight, fltWidth, strideRows, strideCols);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::MaxPoolLayerBackward(TCudaMatrix<AFloat> & activationGradientsBackward,
                                         const TCudaMatrix<AFloat> & activationGradients,
                                         const TCudaMatrix<AFloat> & indexMatrix,
                                         size_t imgHeight,
                                         size_t imgWidth,
                                         size_t fltHeight,
                                         size_t fltWidth,
                                         size_t strideRows,
                                         size_t strideCols,
                                         size_t /* nLocalViews */)
{
   size_t depth = activationGradientsBackward.GetNrows();

   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(activationGradientsBackward);
   hipStream_t s = activationGradientsBackward.GetComputeStream();

   ::TMVA::DNN::Cuda::MaxPoolBackward<<<gridDims, blockDims, 0, s>>>(activationGradientsBackward.GetDataPointer(),
                                                                     activationGradients.GetDataPointer(),
                                                                     indexMatrix.GetDataPointer(),
                                                                     depth, imgHeight, imgWidth, fltHeight, fltWidth,
                                                                     strideRows, strideCols);
}

//____________________________________________________________________________
template<typename AFloat>
void TCuda<AFloat>::Reshape(TCudaMatrix<AFloat> &A, const TCudaMatrix<AFloat> &B)
{
    dim3 blockDims = TDevice::BlockDims2D();
    dim3 gridDims  = TDevice::GridDims2D(A);
    hipStream_t s = A.GetComputeStream();

    ::TMVA::DNN::Cuda::Reshape<<<gridDims, blockDims>>>(A.GetDataPointer(), B.GetDataPointer(),
                                                        A.GetNrows(), A.GetNcols(), B.GetNrows(), B.GetNcols());
}

//______________________________________________________________________________
template <typename AReal>
void TCuda<AReal>::Rearrange(std::vector<TCudaMatrix<AReal>> &out, const std::vector<TCudaMatrix<AReal>> &in)
{
   // B x T x D out --- T x B x D in*/
   size_t B = out.size();
   size_t T = out[0].GetNrows();
   size_t D = out[0].GetNcols();
   if ((T != in.size()) || (B != in[0].GetNrows()) 
       || (D != in[0].GetNcols())) {
      std::cout << "Incompatible Dimensions\n"
         << in.size() << "x" << in[0].GetNrows() << "x" << in[0].GetNcols() 
         << " --> " << B << "x" << T << "x" << D << "\n";
      return;
   }
   for (size_t i = 0; i < B; ++i) {
      for (size_t j = 0; j < T; ++j) {
         for (size_t k = 0; k < D; ++k) {
            out[i](j, k) = in[j](i, k);
         }
      }
   }
   return;
}

//____________________________________________________________________________
////////////////////////////////////////////////////////////////////////////////
/// \brief Flatten a vector of matrices into a single matrix.
///
/// \param[out] A Output matrix.
/// \param[in] B Input vector. Each element is a matrix to be concatenated.
/// \param[in] size Number of matrices in the input vector.
/// \param[in] nRows Number of rows in each matrix of the input vector.
/// \param[in] nCols Number of columns on each matrix of the input vector.
///
/// Each row in the output matrix is the concatenation of the same row in
/// each of the input matrices. Passing an std::vector to a CUDA kernel is
/// a non trivial task that requires manually allocating and copying to device
/// memory - details in comments within the function's body. Launching one
/// thread per output element.
//////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
void TCuda<AFloat>::Flatten(TCudaMatrix<AFloat> &A,
                            const std::vector<TCudaMatrix<AFloat>> &B,
                            size_t size,
                            size_t nRows,
                            size_t nCols)
{
   dim3 blockDims = TDevice::BlockDims2D();
   dim3 gridDims  = TDevice::GridDims2D(A);
   hipStream_t s = A.GetComputeStream();

   // Get raw pointers from a vector of matrices - this is more challenging than it sounds.
   //
   // Attention: While `TCudaMatrix.GetDataPointer() returns a pointer to device memory,
   //            std::vector (and its .data() raw pointer) resides on host memory. Therefore
   //            we need to manually copy these pointers to the device prior to invoking the kernel.

   const AFloat ** dB; // device pointer to device pointers.S
   const AFloat ** hB = new const AFloat * [size]; // host pointer to device pointers.

   hipMalloc(&dB, sizeof(AFloat *) * size);

   for(size_t i = 0; i < size; ++i) {
      hB[i] = B[i].GetDataPointer();
   }

   hipMemcpy(dB, hB, sizeof(AFloat *) * size, hipMemcpyHostToDevice);

   // Launch the kernel using our device pointers.
   ::TMVA::DNN::Cuda::Flatten<<<gridDims, blockDims>>>(A.GetDataPointer(), dB, size, nRows, nCols);
}

//____________________________________________________________________________
////////////////////////////////////////////////////////////////////////////////
/// \brief Deflatten a matrix into a vector of matrices.
///
/// \param[out] A Output matrices. Each element will be a part of the input.
/// \param[in] B Input flat matrix.
/// \param[in] size Number of matrices in the output vector.
/// \param[in] nRows Number of rows in each matrix of the output vector.
/// \param[in] nCols Number of columns on each matrix of the output vector.
///
/// Each row in the input matrix is the concatenation of the same row in
/// each of the output matrices. Passing an std::vector to a CUDA kernel is
/// a non trivial task that requires manually allocating and copying to device
/// memory - details in comments within the function's body. Launching one
/// thread per input element.
//////////////////////////////////////////////////////////////////////////////////
template<typename AFloat>
void TCuda<AFloat>::Deflatten(std::vector<TCudaMatrix<AFloat>> &A,
                              const TCudaMatrix<AFloat> &B,
                              size_t size,
                              size_t nRows,
                              size_t nCols)
{
    dim3 blockDims = TDevice::BlockDims2D();
    dim3 gridDims  = TDevice::GridDims2D(B);
    hipStream_t s = B.GetComputeStream();

    // Get raw pointers from a vector of matrices - this is more challenging than it sounds.
    //
    // Attention: While `TCudaMatrix.GetDataPointer() returns a pointer to device memory,
    //            std::vector (and its .data() raw pointer) resides on host memory. Therefore
    //            we need to manually copy these pointers to the device prior to invoking the kernel.

    AFloat ** dA; // device pointer to device pointers.
    AFloat ** hA = new AFloat * [size]; // host pointer to device pointers.

    hipMalloc(&dA, sizeof(AFloat *) * size);

    for(size_t i = 0; i < size; ++i) {
        hA[i] = A[i].GetDataPointer();
    }

    hipMemcpy(dA, hA, sizeof(AFloat *) * size, hipMemcpyHostToDevice);

    // Launch the kernel using our device pointers.
    ::TMVA::DNN::Cuda::Deflatten<<<gridDims, blockDims>>>(dA, B.GetDataPointer(), size, nRows, nCols);
}

} // namespace DNN
} // namespace TMVA
